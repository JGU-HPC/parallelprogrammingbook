#include "hip/hip_runtime.h"
#include "../include/hpc_helpers.hpp"
#include "../include/binary_IO.hpp"
#include "../include/bitmap_IO.hpp"

template <
    typename index_t,
    typename value_t> __global__
void compute_mean_kernel(
    value_t * Data,
    value_t * Mean,
    index_t num_entries,
    index_t num_features);

int main (int argc, char * argv[]) {

    // set the identifier of the used CUDA device
    hipSetDevice(0);

    // 202599 grayscale images each of shape 55 x 45
    constexpr uint64_t imgs = 202599, rows = 55, cols = 45;

    // pointer for data matrix and mean vector
    float * data = nullptr, * mean = nullptr;
    hipHostMalloc(&data, sizeof(float)*imgs*rows*cols);                  CUERR
    hipHostMalloc(&mean, sizeof(float)*rows*cols);                       CUERR

    // allocate storage on GPU
    float * Data = nullptr, * Mean = nullptr;
    hipMalloc(&Data, sizeof(float)*imgs*rows*cols);                      CUERR
    hipMalloc(&Mean, sizeof(float)*rows*cols);                           CUERR

    // load data matrix from disk
    TIMERSTART(read_data_from_disk)
    std::string file_name = "./data/celebA_gray_lowres.202599_55_45_32.bin";
    load_binary(data, imgs*rows*cols, file_name);
    TIMERSTOP(read_data_from_disk)

    // copy data to device and reset Mean
    TIMERSTART(data_H2D)
    hipMemcpy(Data, data, sizeof(float)*imgs*rows*cols,
               hipMemcpyHostToDevice);                                   CUERR
    hipMemset(Mean, 0, sizeof(float)*rows*cols);                         CUERR
    TIMERSTOP(data_H2D)

    // compute mean
    TIMERSTART(compute_mean_kernel)
    compute_mean_kernel<<<SDIV(rows*cols, 32), 32>>>
                       (Data, Mean, imgs, rows*cols);                     CUERR
    TIMERSTOP(compute_mean_kernel)


    // transfer mean back to host
    TIMERSTART(mean_D2H)
    hipMemcpy(mean, Mean, sizeof(float)*rows*cols,
               hipMemcpyDeviceToHost);                                   CUERR
    TIMERSTOP(mean_D2H)

    // write mean image to disk
    TIMERSTART(write_mean_image_to_disk)
    dump_bitmap(mean, rows, cols, "./imgs/celebA_mean.bmp");
    TIMERSTOP(write_mean_image_to_disk)

    // get rid of the memory
    hipHostFree(data);                                                   CUERR
    hipHostFree(mean);                                                   CUERR
    hipFree(Data);                                                       CUERR
    hipFree(Mean);                                                       CUERR

}

template <
    typename index_t,
    typename value_t> __global__
void compute_mean_kernel(
    value_t * Data,
    value_t * Mean,
    index_t num_entries,
    index_t num_features) {

    auto thid = blockDim.x*blockIdx.x + threadIdx.x;

    if (thid < num_features) {

        value_t accum = 0;

        # pragma unroll 32
        for (index_t entry = 0; entry < num_entries; entry++)
            accum += Data[entry*num_features+thid];

        Mean[thid] = accum/num_entries;
    }
}

